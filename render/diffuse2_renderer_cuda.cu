#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2015-2018 Intel Corporation                                    //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include <stdexcept>
#include "core/intersector_cuda.cuh"
#include "geometry/triangle_mesh_cuda.cuh"
#include "sampling/random_sampler.cuh"
#include "camera/pinhole_camera_cuda.cuh"
#include "camera/thin_lens_camera_cuda.cuh"
#include "diffuse2_renderer_cuda.h"

namespace prt {

template <class CameraCuda>
static CUDA_DEV_KERNEL void generateRaysKernel(CameraCuda camera,
                                               AccumBufferCuda accumBuffer,
                                               int pass,
                                               RayCuda* rays,
                                               int* pixelIds,
                                               unsigned int* samplerStates,
                                               float* L)
{
    // Generate rays in Morton order
    int tx = threadIdx.x;
    int mx = (tx & 1) | ((tx & 4) >> 1) | ((tx & 16) >> 2);
    int my = ((tx & 2) >> 1) | ((tx & 8) >> 2);
    int x = blockIdx.x * 8 + mx;
    int y = blockIdx.y * 16 + threadIdx.y * 4 + my;

    int pixelId = x + y * accumBuffer.size.x;

    RandomSampler sampler;
    sampler.init(pass, pixelId);

    CameraSampleCuda cameraSample;
    float2 pixelSample = sampler.get2D();
    cameraSample.image = (make_float2(x, y) + pixelSample) / make_float2(accumBuffer.size.x, accumBuffer.size.y);
    cameraSample.lens = sampler.get2D();

    RayCuda ray;
    getRay(camera, ray, cameraSample);

    int i = (blockDim.x * blockDim.y) * (blockIdx.y * gridDim.x + blockIdx.x) + (threadIdx.y * blockDim.x + threadIdx.x);
    rays[i] = ray;
    pixelIds[i] = pixelId;
    samplerStates[i] = sampler.getState();
    L[i] = 0.f;
}

template <class ShadingContextT, bool isAccum>
static CUDA_DEV_KERNEL void shadeRaysKernel(TriangleMeshCuda mesh,
                                            AccumBufferCuda accumBuffer,
                                            const RayCuda* rays, RayCuda* rays_o,
                                            const HitCuda* hits,
                                            RayCuda* shadowRays_o,
                                            const AnyHitCuda* shadowHits,
                                            const int* pixelIds, int* pixelIds_o,
                                            const unsigned int* samplerStates, unsigned int* samplerStates_o,
                                            const float* L, float* L_o,
                                            float throughput,
                                            int* queueSize,
                                            int count,
                                            bool final)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= count)
        return;

    float color = L[i];
    if (shadowHits && !shadowHits[i].isHit())
        color += throughput * 0.5f; // with MIS weight

    HitCuda hit = hits[i];
    int pixelId = pixelIds[i];

    bool isHit = hit.isHit();

    if (isHit)
    {
        if (!final)
        {
            int o = atomicIncAgg(queueSize);

            RayCuda ray = rays[i];
            ShadingContextT ctx;
            postIntersect(mesh, ray, hit, ctx);

            RandomSampler sampler;
            sampler.init(samplerStates[i]);

            // Generate shadow ray
            float2 s = sampler.get2D();
            ray.init(ctx.p, ctx.getFrame() * cosineSampleHemisphere(s), ctx.eps);
            shadowRays_o[o] = ray;

            // Generate extension ray
            s = sampler.get2D();
            ray.init(ctx.p, ctx.getFrame() * cosineSampleHemisphere(s), ctx.eps);
            rays_o[o] = ray;

            pixelIds_o[o] = pixelId;
            samplerStates_o[o] = sampler.getState();
            L_o[o] = color;
        }
    }
    else
    {
        color += throughput * (shadowHits ? 0.5f : 1.f); // with MIS weight
    }

    if (!isHit || final)
    {
        if (isAccum)
        {
            float4 accum = accumBuffer.data[pixelId];
            accumBuffer.data[pixelId] = make_float4(color+accum.x, color+accum.y, color+accum.z, 1.0f+accum.w);
        }
        else
        {
            accumBuffer.data[pixelId] = make_float4(color, color, color, 1.0f);
        }
    }
}

Diffuse2RendererCuda::Diffuse2RendererCuda(const TriangleMeshCuda& mesh, IntersectorStreamCuda* intersector, int imageSize, int maxDepth, bool isFast)
    : mesh(mesh),
      intersector(intersector),
      pixelCount(imageSize),
      pass(0),
      maxDepth(maxDepth),
      isFast(isFast)
{
    hipMalloc(&hits, imageSize * sizeof(HitCuda));
    hipMalloc(&shadowRays, imageSize * sizeof(RayCuda));
    hipMalloc(&shadowHits, imageSize * sizeof(AnyHitCuda));
    for (int i = 0; i < 2; ++i)
    {
        hipMalloc(&rays[i], imageSize * sizeof(RayCuda));
        hipMalloc(&pixelIds[i], imageSize * sizeof(int));
        hipMalloc(&samplerStates[i], imageSize * sizeof(int));
        hipMalloc(&L[i], imageSize * sizeof(float));
    }
    hipMalloc(&queueSize, sizeof(int));

    hipFuncSetAttribute(reinterpret_cast<const void*>(generateRaysKernel<PinholeCameraCuda>),            hipFuncAttributePreferredSharedMemoryCarveout, cudaSharedmemCarveoutMaxL1);
    hipFuncSetAttribute(reinterpret_cast<const void*>(generateRaysKernel<ThinLensCameraCuda>),           hipFuncAttributePreferredSharedMemoryCarveout, cudaSharedmemCarveoutMaxL1);
    hipFuncSetAttribute(reinterpret_cast<const void*>(shadeRaysKernel<ShadingContextCuda), true>,        hipFuncAttributePreferredSharedMemoryCarveout, cudaSharedmemCarveoutMaxL1);
    hipFuncSetAttribute(reinterpret_cast<const void*>(shadeRaysKernel<SimpleShadingContextCuda), false>, hipFuncAttributePreferredSharedMemoryCarveout, cudaSharedmemCarveoutMaxL1);
}

Diffuse2RendererCuda::~Diffuse2RendererCuda()
{
    hipFree(hits);
    hipFree(shadowRays);
    hipFree(shadowHits);
    for (int i = 0; i < 2; ++i)
    {
        hipFree(rays[i]);
        hipFree(pixelIds[i]);
        hipFree(samplerStates[i]);
        hipFree(L[i]);
    }
    hipFree(queueSize);

    delete intersector;
}

int Diffuse2RendererCuda::render(const ThinLensCameraCuda& camera, const AccumBufferCuda& accumBuffer)
{
    if (accumBuffer.size.x % 8 != 0 || accumBuffer.size.y % 16 != 0)
        throw std::logic_error("image size is not divisible by the tile size");

    int totalRayCount = 0;
    int count = pixelCount;
    int buf = 0;

    // Generate camera rays
    dim3 genBlockSize(32, 4);
    dim3 genGridSize(accumBuffer.size.x / 8, accumBuffer.size.y / 16);
    if (camera.lensRadius == 0.0f)
        generateRaysKernel<<<genGridSize, genBlockSize>>>((const PinholeCameraCuda&)camera, accumBuffer, pass, rays[buf], pixelIds[buf], samplerStates[buf], L[buf]);
    else
        generateRaysKernel<<<genGridSize, genBlockSize>>>(camera, accumBuffer, pass, rays[buf], pixelIds[buf], samplerStates[buf], L[buf]);

    dim3 blockSize(256);
    dim3 gridSize((count + blockSize.x - 1) / blockSize.x);

    float throughput = 1.f;

    int depth = 0;
    while (count > 0)
    {
        // Intersect the rays
        intersector->intersect(rays[buf], hits, count);
        totalRayCount += count;
        if (depth > 0)
        {
            intersector->occluded(shadowRays, shadowHits, count);
            totalRayCount += count;
        }

        // Shade the rays
        int emptyQueueSize = 0;
        hipMemcpy(queueSize, &emptyQueueSize, sizeof(int), hipMemcpyHostToDevice);

        int buf2 = 1-buf;
        bool final = depth == maxDepth;

        if (isFast)
            shadeRaysKernel<SimpleShadingContextCuda, false><<<gridSize, blockSize>>>(mesh, accumBuffer, rays[buf], rays[buf2], hits, shadowRays, (depth > 0) ? shadowHits : nullptr, pixelIds[buf], pixelIds[buf2], samplerStates[buf], samplerStates[buf2], L[buf], L[buf2], throughput, queueSize, count, final);
        else
            shadeRaysKernel<ShadingContextCuda, true><<<gridSize, blockSize>>>(mesh, accumBuffer, rays[buf], rays[buf2], hits, shadowRays, (depth > 0) ? shadowHits : nullptr, pixelIds[buf], pixelIds[buf2], samplerStates[buf], samplerStates[buf2], L[buf], L[buf2], throughput, queueSize, count, final);

        if (final)
            break;

        throughput *= 0.8f;

        hipMemcpy(&count, queueSize, sizeof(int), hipMemcpyDeviceToHost);
        buf = buf2;
        depth++;
    }

    pass++;
    return totalRayCount;
}

} // namespace prt
